#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This example demonstrates a simple vector sum on the GPU and on the host.
 * sumArraysOnGPU splits the work of the vector sum across CUDA threads on the
 * GPU. Only a single thread block is used in this small case, for simplicity.
 * sumArraysOnHost sequentially iterates through vector elements on the host.
 */

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
                   gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match.\n\n");

    return;
}


void initialData(float *ip, int size)
{
    // generate different seed for random number
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }

    return;
}


void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
        C[idx] = A[idx] + B[idx];
}

//注意这里的i是基于threadIdx.x得到的，所以一个block的每个线程虽然执行的任务形式一样
//但由于idx不一样，所以i是不一样的。
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = threadIdx.x;

    if (i < N) C[i] = A[i] + B[i];
    //发现在GPU的核函数里也可以调用printf啊，神奇。
    //所以GPU里的数据也不是显示不了。
    //接下来要看一下CPU上能不能显示GPU的数据
    printf("IN GPU %d:%f\n",i,C[i]);
}


int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);
    //设置一下GPU
    // set up device
    int dev = 0;
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int nElem = 1 << 5;
    printf("Vector size %d\n", nElem);
    //分配CPU内存(host开头)
    // malloc host memory
    size_t nBytes = nElem * sizeof(float);
    
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A     = (float *)malloc(nBytes);
    h_B     = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);

    // initialize data at host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);

    memset(hostRef, 0, nBytes);
    memset(gpuRef,  0, nBytes);

    // malloc device global memory
    //分配GPU内存（d开头），这个是host，所以应该也是同步的。
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));
    //把数据从内存迁移到显存。这个是同步的。
    // transfer data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    dim3 block (nElem);
    dim3 grid  (1);
    //1个block，block里有32个线程
    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
    printf("Execution configure <<<%d, %d>>>\n", grid.x, block.x);

    // copy kernel result back to host side
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    for(int i=0;i<32;i++){
        printf("CPU %d:%f\n",i,gpuRef[i]);
    }
    // //会提示Segmentation fault (core dumped)
    // for(int i=0;i<32;i++){
    //     printf("CPU %d:%f",i,d_C[i]);
    // }
    // add vector at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    CHECK(hipDeviceReset());
    return(0);
}
