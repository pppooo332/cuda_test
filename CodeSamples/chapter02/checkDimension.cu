#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Display the dimensionality of a thread block and grid from the host and
 * device.
 */
//
//对于每一个线程而言，blockDim都是一样的，因为blockDim表示这个线程所在的这个线程块的结构。
//threadIdx则表示这个线程在这个线程块中的位置，所以对于每个线程都不一样（从0开始。）
//gridDim表示这个线程所在的网格的整体结构
//blockIdx表示这个线程所在的线程块在网格中的位置。
//所以说我们会发现由于网格中有多个线程块，单凭threadIdx是无法唯一表示一个线程的。
//只有threadIdx+blockIdx才能唯一表示一个线程，这点还是挺复杂的。
__global__ void checkIndex(void)
{
    printf("threadIdx:(%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z);
    printf("blockIdx:(%d, %d, %d)\n", blockIdx.x, blockIdx.y, blockIdx.z);

    printf("blockDim:(%d, %d, %d)\n", blockDim.x, blockDim.y, blockDim.z);
    printf("gridDim:(%d, %d, %d)\n", gridDim.x, gridDim.y, gridDim.z);

}

int main(int argc, char **argv)
{
    // define total data element
    int nElem = 6;

    // define grid and block structure
    //每个线程块有3个线程。
    dim3 block(3);
    //一共有8/3=2个线程块。
    dim3 grid((nElem + block.x - 1) / block.x);

    // check grid and block dimension from host side
    //看来host来看的话，y,z是默认都初始化为1。
    printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
    printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);

    // check grid and block dimension from device side
    checkIndex<<<grid, block>>>();

    // reset device before you leave
    CHECK(hipDeviceReset());

    return(0);
}
