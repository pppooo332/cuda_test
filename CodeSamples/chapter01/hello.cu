#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <unistd.h>

/*
 * A simple introduction to programming in CUDA. This program prints "Hello
 * World from GPU! from 10 CUDA threads running on the GPU.
 */
//修饰符__global__告诉编译器这个函数将会从CPU中调用，然后在GPU上执行。
__global__ void helloFromGPU()
{
    printf("Hello World from GPU!\n ");
}

int main(int argc, char **argv)
{
    printf("Hello World from CPU!\n thread %d",threadIdx.x);
    //10个线程执行这个函数
    helloFromGPU<<<1, 10>>>();
    //用来显式地释放和清空当前进程中与当前设备有关的所有资源。
    //注释掉这个之后，GPU执行不了print，难道是因为这个函数也有阻塞的功能，让主线程不要过早结束？
    //果然，加上sleep之后就可以输出了。
    //sleep(2);
    CHECK(hipDeviceReset());
    return 0;
}
//nvcc -o hello hello.cu
//./hello

